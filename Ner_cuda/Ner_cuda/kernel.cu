#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
//#include "../../core/network.hpp"
#include <iostream>
#include <vector>

/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
*/

enum TrainingAlgorithm {
	StochasticGradient = 0,
	Adagrad,
	RMSProp,
	Adam
};

__device__ double transferFunction(double x)
{
	return 1.0 / (1.0 + std::exp(-x));
}

__device__ double transferFunctionDerivative(double x)
{
	return (1.0 - x) * x;
}

__global__ void forwardKernel(double *outputs, double *weightes, const unsigned layer, const unsigned inputs, const unsigned outputs_size, const unsigned layers, const unsigned neurons)
{
	int i = threadIdx.x;

	unsigned neurons_size = layer == layers + 1 ? outputs_size : neurons;
	unsigned offset_neuron = inputs + 1 + (layer - 1) * (neurons + 1);
	unsigned prev_layer_size = (layer == 1 ? inputs : neurons) + 1;
	unsigned prev_layer_offset_neuron = offset_neuron - prev_layer_size;
	unsigned prev_layer_weight_offset = (layer == 1 ? 0 : (inputs + 1) * neurons);
	if (layer > 1)
		prev_layer_weight_offset += (neurons + 1) * neurons * (layer - 2);

	// prev layer
	double sum = 0;
	for (unsigned j = 0; j < prev_layer_size; ++j)
	{
		sum += outputs[prev_layer_offset_neuron + j] * weightes[prev_layer_weight_offset + j * neurons_size + i];
	}
	outputs[offset_neuron + i] = transferFunction(sum);

	printf("out = %f\n", outputs[i + offset_neuron]);
}

__global__ void calculateOutputDelta(double *outputs, double *delta, double *targets, const unsigned outputs_offset)
{
	int i = threadIdx.x;

	double delta_ = targets[i] - outputs[i + outputs_offset];
	delta[i + outputs_offset] = delta_ * transferFunctionDerivative(outputs[i + outputs_offset]);
}


__global__ void calculateHiddensDelta(double *outputs, double *weightes, double *delta, const unsigned layer, const unsigned inputs, const unsigned outputs_size, const unsigned layers, const unsigned neurons)
{
	int i = threadIdx.x;

	unsigned neurons_size = neurons + 1;
	unsigned offset_neuron = inputs + 1 + (layer - 1) * (neurons + 1);
	unsigned weight_offset = (inputs + 1) * neurons + (neurons + 1) * neurons * (layer - 1);
	unsigned next_layer_size = (layer == layers ? outputs_size : neurons);
	unsigned next_layer_offset_neuron = offset_neuron + neurons_size;

	double dow = 0.;
	for (unsigned n = 0; n < next_layer_size; ++n)
	{
		dow += weightes[weight_offset + i * next_layer_size + n] * delta[next_layer_offset_neuron + n];
	}
	delta[i + offset_neuron] = dow * transferFunctionDerivative(outputs[i + offset_neuron]);
}

__constant__ const double rate = 0.7;
__constant__ const double momentum = 0.3;
__constant__ const double beta1 = 0.9;
__constant__ const double beta2 = 0.999;
__constant__ const double d_epsilon = 0.0000001;

__global__ void updateInputWeights(
	double *outputs,
	double *weightes, 
	double *delta, 
	double *delta_weight, 
	
	const unsigned layer, 
	const unsigned inputs, 
	const unsigned outputs_size, 
	const unsigned layers, 
	const unsigned neurons, 
	
	TrainingAlgorithm algorithm,

	double *algorithm_e,
	double *algorithm_m,
	double *algorithm_v,
	double *algorithm_t
)
{
	int i = threadIdx.x;

	unsigned neurons_size = layer == layers + 1 ? outputs_size : neurons;
	unsigned offset_neuron = inputs + 1 + (layer - 1) * (neurons + 1);
	unsigned prev_layer_size = (layer == 1 ? inputs : neurons) + 1;
	unsigned prev_layer_offset_neuron = offset_neuron - prev_layer_size;
	unsigned prev_layer_weight_offset = (layer == 1 ? 0 : (inputs + 1) * neurons);
	if (layer > 1)
		prev_layer_weight_offset += (neurons + 1) * neurons * (layer - 2);

	for (unsigned j = 0; j < prev_layer_size; ++j)
	{
		unsigned prev_layer_weight_index = prev_layer_weight_offset + j * neurons_size + i;
		double oldDeltaWeight = delta_weight[prev_layer_weight_index];
		double gradient = outputs[prev_layer_offset_neuron + j] * delta[offset_neuron + i];
		double newDeltaWeight;

		switch (algorithm)
		{
		case StochasticGradient:
		{
			newDeltaWeight = rate * gradient + momentum * oldDeltaWeight;

			break;
		}
		case Adagrad:
		{
			double& e = algorithm_e[prev_layer_weight_index];

			e = e + pow(gradient, 2);
			newDeltaWeight = rate * gradient / sqrt(e + d_epsilon);

			break;
		}
		case RMSProp:
		{
			double& e = algorithm_e[prev_layer_weight_index];

			e = momentum * e + (1 - momentum) * pow(gradient, 2);
			newDeltaWeight = rate * gradient / sqrt(e + d_epsilon);

			break;
		}
		case Adam:
		{
			double& m = algorithm_m[prev_layer_weight_index];
			double& v = algorithm_v[prev_layer_weight_index];
			double& t = algorithm_t[prev_layer_weight_index];

			m = beta1 * m + (1 - beta1) * gradient;
			v = beta2 * v + (1 - beta2) * pow(gradient, 2);

			double mt = m / (1 - pow(beta1, t));
			double mv = v / (1 - pow(beta2, t));
			t++;

			newDeltaWeight = rate * mt / sqrt(mv + d_epsilon);

			break;
		}
		default:
			break;
		}

		delta_weight[prev_layer_weight_index] = newDeltaWeight;
		weightes[prev_layer_weight_index] += newDeltaWeight;
	}
}

void forward(double* neuron_outputs, double* neuron_weigths, unsigned inputs, unsigned outputs, unsigned layers, unsigned neurons)
{
	// forward
	for (unsigned layer = 1; layer <= layers + 1; ++layer)
	{
		unsigned threads = (layer == layers + 1) ? outputs : neurons;
		forwardKernel << <1, threads >> >(neuron_outputs, neuron_weigths, layer, inputs, outputs, layers, neurons);
		hipDeviceSynchronize();
	}

}

double error(double* neuron_outputs, double* neuron_targets, unsigned outputs, unsigned outputs_offset_neurons)
{
	double error = 0.0;
	for (unsigned i = 0; i < outputs; ++i)
	{
		double delta = neuron_targets[i] - neuron_outputs[outputs_offset_neurons + i];
		error += delta * delta;
	}
	error /= outputs;
	std::cout << error * 100 << '%' << std::endl;
	return error;
}

void backPropagation(
	double* neuron_outputs, 
	double* neuron_weigths, 
	double* neuron_delta, 
	double* neuron_delta_weight, 
	double* neuron_targets, 
	
	unsigned inputs, 
	unsigned outputs, 
	unsigned layers,
	unsigned neurons, 
	unsigned outputs_offset_neurons,

	TrainingAlgorithm algorithm,

	double *algorithm_e,
	double *algorithm_m,
	double *algorithm_v,
	double *algorithm_t
)
{
	// calculate output delta
	calculateOutputDelta << <1, outputs >> >(neuron_outputs, neuron_delta, neuron_targets, outputs_offset_neurons);
	hipDeviceSynchronize();

	// calculate hidden deltas
	for (unsigned layer = layers; layer > 0; --layer)
	{
		calculateHiddensDelta << <1, neurons + 1 >> >(neuron_outputs, neuron_weigths, neuron_delta, layer, inputs, outputs, layers, neurons);
		hipDeviceSynchronize();
	}

	// update weights
	for (unsigned layer = layers + 1; layer > 0; --layer)
	{
		unsigned threads = (layer == layers + 1) ? outputs : neurons;
		updateInputWeights << <1, threads >> >(
			neuron_outputs, 
			neuron_weigths, 
			neuron_delta, 
			neuron_delta_weight, 
			
			layer, 
			inputs, 
			outputs, 
			layers, 
			neurons,
			
			algorithm,

			algorithm_e,
			algorithm_m,
			algorithm_v,
			algorithm_t
		);
		hipDeviceSynchronize();
	}
}

double randomWeight(void) { return rand() / double(RAND_MAX); }

struct NeuronNetwork
{
	int inputs;
	int outputs;
	int layers;
	int neurons;
	TrainingAlgorithm algorithm = StochasticGradient;

	double* neuron_outputs;
	double* neuron_delta;
	double* neuron_weigths;
	double* neuron_delta_weight;
	double* neuron_targets;

	double* algorithm_e;
	double* algorithm_m;
	double* algorithm_v;
	double* algorithm_t;

	unsigned neurons_size;
	unsigned hidden_offset_neurons;
	unsigned outputs_offset_neurons;
	unsigned neuron_weigths_size;

	NeuronNetwork(int inputs_, int outputs_, int layers_, int neurons_, TrainingAlgorithm algorithm_ = StochasticGradient)
	{
		algorithm = algorithm_;

		inputs = inputs_;
		outputs = outputs_;
		layers = layers_;
		neurons = neurons_;

		neurons_size = inputs + 1 + outputs + (neurons + 1) * layers;
		hidden_offset_neurons = inputs + 1;
		outputs_offset_neurons = hidden_offset_neurons + (neurons + 1) * layers;
		neuron_weigths_size = ((neurons + 1) * neurons) * (layers - 1) + ((inputs + 1) * neurons) + (outputs * (neurons + 1));

		hipMallocManaged(&neuron_outputs, neurons_size * sizeof(double));
		hipMallocManaged(&neuron_delta, neurons_size * sizeof(double));
		hipMallocManaged(&neuron_weigths, neuron_weigths_size * sizeof(double));
		hipMallocManaged(&neuron_delta_weight, neuron_weigths_size * sizeof(double));
		hipMallocManaged(&neuron_targets, outputs * sizeof(double));
		hipMallocManaged(&algorithm_e, neuron_weigths_size * sizeof(double));
		hipMallocManaged(&algorithm_m, neuron_weigths_size * sizeof(double));
		hipMallocManaged(&algorithm_v, neuron_weigths_size * sizeof(double));
		hipMallocManaged(&algorithm_t, neuron_weigths_size * sizeof(double));

		// bias neurons
		for (unsigned layer = 0, i = 0; layer < layers + 1; ++layer)
		{
			unsigned layer_size = layer == 0 ? inputs + 1 : neurons + 1;
			i += layer_size;
			neuron_outputs[i - 1] = 1;
		}

		// first t = 1
		for (unsigned i = 0; i < neuron_weigths_size; ++i)
		{
			algorithm_t[i] = 1;
		}

		// random weightes
		for (unsigned i = 0; i < neuron_weigths_size; ++i)
		{
			neuron_weigths[i] = randomWeight();
		}
	}

	~NeuronNetwork() 
	{
		hipFree(neuron_outputs);
		hipFree(neuron_delta);
		hipFree(neuron_weigths);
		hipFree(neuron_delta_weight);
		hipFree(neuron_targets);

		hipFree(algorithm_e);
		hipFree(algorithm_m);
		hipFree(algorithm_v);
		hipFree(algorithm_t);

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		hipDeviceReset();
	}

	void train(double* i, double* o)
	{
		memcpy(neuron_outputs, i, sizeof(double) * inputs);
		memcpy(neuron_targets, o, sizeof(double) * outputs);
		forward(neuron_outputs, neuron_weigths, inputs, outputs, layers, neurons);
		error(neuron_outputs, neuron_targets, outputs, outputs_offset_neurons);
		backPropagation(
			neuron_outputs,
			neuron_weigths,
			neuron_delta,
			neuron_delta_weight,
			neuron_targets,

			inputs,
			outputs,
			layers,
			neurons,
			outputs_offset_neurons,

			algorithm,

			algorithm_e,
			algorithm_m,
			algorithm_v,
			algorithm_t
		);
	}

	void train(std::vector<double> i, std::vector<double> o)
	{
		train(i.data(), o.data());
	}
};

int main()
{
	NeuronNetwork n(2, 1, 1, 2, StochasticGradient);

	n.neuron_weigths[0] = 0.45;
	n.neuron_weigths[1] = 0.78;
	n.neuron_weigths[2] = -0.12;
	n.neuron_weigths[3] = 0.13;
	n.neuron_weigths[6] = 1.5;
	n.neuron_weigths[7] = -2.3;

	n.train(std::vector<double>{ 1, 0 }, std::vector<double>{ 1 });
	n.train(std::vector<double>{ 1, 0 }, std::vector<double>{ 1 });
	

    return 0;
}
